#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

__global__ void test_random(double* rand_d, int length, hiprandState* s){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<length){
        hiprand_init(0,id,0,&s[id]);
        rand_d[id] = hiprand_uniform_double(&s[id]);
    }
}

int main(void)
{
    hipSetDeviceFlags(hipDeviceMapHost);
    int length=100000;

    double* data_d;
    double* data_h;
    hipMalloc(&data_d, length*sizeof(double));
    data_h =  (double*)malloc(length*sizeof(double));

	hiprandState *s;
    hipMalloc(&s,length*sizeof(hiprandState));

    int const n_thread = length/200;
    int const n_block = 200;
    test_random<<<n_block,n_thread>>>(data_d,length,s);
    hipError_t err;
    err = hipDeviceSynchronize();

    hipMemcpy(data_h,data_d,length*sizeof(double),hipMemcpyDeviceToHost);

    for(int i=0;i<length;++i) printf("%.5f \n",data_h[i]);

    hipFree(data_d);
    free(data_h);
    hipFree(s);
}
